#include "hip/hip_runtime.h"
//-*-c++-*-
//////////////////////////////////////////////////////////////////
#include "stdio.h"
#include "cudainfo.h"
#include "nnet.h"
#include "kernel_dims.h"


template <typename cuda_type>
struct sigmoid_functor 
{
  __device__ __forceinline__ cuda_type operator ()( cuda_type val)
  {
    return 1.0f/(1 + exp(-val));
  }
};

template <typename cuda_type>
struct linear_functor 
{
  __device__ __forceinline__ cuda_type operator () ( cuda_type val)
  {
    //Needs to be a better way to do this
    return val > 0.0f ? 1.0f : 0.0f;
  }
};

///////////////////////////////////////////////////////////
// nnets are small. So

/*template <typename cuda_type, typename activ_type >
  __global__ void cu_compute_layer_kernel(cuda_type *X, cuda_type *W,  
  cuda_type *Y, size_t inputs, size_t outputs,
  activ_type activator = activ_type()) 
  {
  // read in the weights
  }*/


/*template <typename cuda_type, typename activ_type >
  __global__ void cu_compute_layer_kernel(cuda_type *X, cuda_type *W,  
  cuda_type *Y, size_t inputs, size_t outputs,
  activ_type activator = activ_type()) 
  {

  unsigned int tx = threadIdx.x;
  unsigned int netid = tx / outputs;
  unsigned int yid = tx % outputs;
  unsigned int netsize = (inputs + 1)*outputs;

  cuda_type value = W[netsize*netid + (inputs + 1)*yid + inputs];
  for (int i=0; i < inputs; ++i)
  {
  value += X[inputs*netid+i]*W[ netsize*netid + (inputs + 1)*yid + i];
  }
  Y[tx] = activator ( value );

  };*/



/////////////////////////////////////////////////////////////////////////////////////
//Description: neural networks layer compute
//What it needs to accomplish:
//I = inputs
//O = outputs
//N = Individuals
//S = start points
//
//Passing in N*S*(I+1)*O weights
//Passing in N*S*I inputs
//Expecting N*S*O outputs
//

//What we expect is that there should only be
// W*I weights in memory ever
// 
//

__shared__ void * compute_layer_shared_mem;
template <typename cuda_type, typename activ_type >
__global__ void cu_compute_layer_kernel(cuda_type *X, cuda_type *W,  
					cuda_type *Y, size_t inputs, size_t outputs,
					size_t tasks_per_block, 
					size_t individuals, size_t points, 
					activ_type activator = activ_type()) 
{

  //cuda_type * s_weights = (cuda_type *) compute_layer_shared_mem; 
  //cuda_type * s_inputs = & ((cuda_type *) compute_layer_shared_mem)[inputs * outputs + 1]; // plus some offset

  size_t tx = blockIdx.x * blockDim.x + threadIdx.x;
  //Add check for last block that will be running less than normal threads
  if (threadIdx.x < tasks_per_block)
    {

      //0. load shared memory with inputs and weights. 
      unsigned int netid = tx / outputs;
      unsigned int yid = tx % outputs;
      unsigned int netsize = (inputs + 1)*outputs;
   
      //1. load in the bias
      cuda_type value = W[netsize*netid + (inputs + 1)*yid + inputs];

      //2. Add the weights * inputs.
      for (int i=0; i < inputs; ++i)
	{
	  value += X[inputs*netid+i]*W[ netsize*netid + (inputs + 1)*yid + i];
	}

      //3. save to output
      Y[netid*outputs+yid] = value;//activator ( value );     
    }
};

template <typename cuda_type>
static void print_parameters(cuda_type *X, cuda_type *W,  cuda_type *Y, size_t inputs, cuda::kernel_dimensions * dims_)
{
  printf("%d\n", dims_);
  printf("cu_compute_layer with \nX = %x\n W = %x\n Y = %x\n inputs = %d,\n task_size = %d\n tasks_per_block = %d\n individuals = %d\n points=%d\n", 
	 (size_t)X, (size_t)W, (size_t)Y, inputs, dims_->get_task_size(), dims_->get_tasks_per_block(), 
	 dims_->get_individuals(), dims_->get_points());
}

template <typename cuda_type, typename activ_type>
__host__ void cu_compute_layer(cuda_type *X, cuda_type *W,  cuda_type *Y, size_t inputs, cuda::kernel_dimensions * dims_)
{
  print_parameters(X, W, Y, inputs, dims_);
  cu_compute_layer_kernel<cuda_type, activ_type> 
    <<<dims_->get_grid_dims(),
    dims_->get_block_dims(), 
    dims_->get_shared_mem_size()>>>
    (X, W, Y, inputs, dims_->get_task_size(), 
     dims_->get_tasks_per_block(), 
     dims_->get_individuals(), 
     dims_->get_points());
}

template <>
__host__ void cu_compute_layer<float, linear_functor<float> >(float *X, float *W,  float *Y, size_t inputs, cuda::kernel_dimensions * dims_)
{
  print_parameters(X, W, Y, inputs, dims_);
  cu_compute_layer_kernel<float, linear_functor <float> >
    <<<dims_->get_grid_dims(),
    dims_->get_block_dims(), 
    dims_->get_shared_mem_size()>>>
    (X, W, Y, inputs, dims_->get_task_size(),
     dims_->get_tasks_per_block(), 
     dims_->get_individuals(),
     dims_->get_points());
}

template <>
__host__ void cu_compute_layer<float, sigmoid_functor<float> > (float *X, float *W,  float *Y, size_t inputs, cuda::kernel_dimensions * dims_)
{

  print_parameters(X, W, Y, inputs, dims_);
  cu_compute_layer_kernel<float, sigmoid_functor<float> >
    <<<dims_->get_grid_dims(),
    dims_->get_block_dims(), 
    dims_->get_shared_mem_size()>>>
    (X, W, Y, inputs, dims_->get_task_size(), 
     dims_->get_tasks_per_block(), 
     dims_->get_individuals(), dims_->get_points());
}

template <>
__host__ void cu_compute_layer<double, linear_functor<double> > (double *X, double *W,  double *Y, size_t inputs, cuda::kernel_dimensions * dims_)
{
  print_parameters(X, W, Y, inputs, dims_);
  cu_compute_layer_kernel<double, linear_functor<double> >
    <<<dims_->get_grid_dims(),
    dims_->get_block_dims(), 
    dims_->get_shared_mem_size()>>>
    (X, W, Y, inputs, dims_->get_task_size(),
     dims_->get_tasks_per_block(), 
     dims_->get_individuals(),
     dims_->get_points());
}

template <>
__host__ void cu_compute_layer<double, sigmoid_functor<double> > (double *X, double *W,  double *Y, 
								  size_t inputs, cuda::kernel_dimensions * dims_)
{
  print_parameters(X, W, Y, inputs, dims_);
  cu_compute_layer_kernel<double, sigmoid_functor<double> >
    <<<dims_->get_grid_dims(),
    dims_->get_block_dims(), 
    dims_->get_shared_mem_size()>>>
    (X, W, Y, inputs, dims_->get_task_size(), 
     dims_->get_tasks_per_block(), 
     dims_->get_individuals(),
     dims_->get_points());
}


///////////////////////////////////////////////////////////
/*template <typename cuda_type, typename activ_type>
  __global__ void cu_compute_layer_with_segments_kernel(cuda_type *X,  cuda_type *W,  cuda_type *Y, int width, int seg,    
  activ_type activator = activ_type()) 
  {

  unsigned int bx = blockIdx.x, by = blockIdx.y;
  unsigned int tx = threadIdx.x, ty = threadIdx.y;

  //The order of weights is as follows:
  //1) the weights between X and Y
  //2) the bias for Y
  //3) the weights for the memory component*
  unsigned int offset = tx*(width+1);

  cuda_type value = W[offset + seg];
  for (unsigned int i=0; i < seg; ++i)
  {
  value += X[i]*W[offset + i];
  }

  for (unsigned int i=seg; i < width; ++i)
  {
  value += X[i]*W[offset +  i  + 1];
  }

  Y[tx] = activator( value );
  }


  template <typename cuda_type, typename activ_type>
  void cu_compute_layer_with_segments(cuda_type *X, cuda_type *W,  
  cuda_type *Y, int width, int seg,
  dim3 gridsize, dim3 blocksize)
  {
  cu_compute_layer_with_segments_kernel<cuda_type, activ_type><<<gridsize, blocksize>>>(X, W, Y, width, seg);
  }*/
