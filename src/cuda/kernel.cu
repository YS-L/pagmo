#include "hip/hip_runtime.h"
//-*-c++-*-
//Basic kernel to compute the result of a layer's operation

#ifndef __PAGMO_CUDA_KERNELS__
#define  __PAGMO_CUDA_KERNELS__

#include "stdio.h"
#include "kernel.h"


template <typename cuda_type>
struct nop_functor 
{
  __device__ __forceinline__ cuda_type operator() ( cuda_type val )
  {
    return val;
  }
};

template <typename cuda_type>
struct scale_functor 
{
  __device__ __forceinline__ cuda_type operator ()(cuda_type val, const cuda_type max_val)
  {
    return ( val - 0.5f )* 2 * max_val;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////
// computes y += alpha * x1
template <typename cuda_type>
__device__ __forceinline__ void increment(cuda_type *Y,  cuda_type *X,  const cuda_type alpha) 
{
  unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  Y[idx] += alpha * X[idx];
}

// computes y = x1 - x2
template <typename cuda_type>
__device__ __forceinline__ void assign_diff(cuda_type *Y,  cuda_type *X1,  cuda_type * X2) 
{
  unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  Y[idx] = X1[idx] - X2[idx];
}

// computes y = x1 + alpha * x2
template <typename cuda_type>
__device__ __forceinline__ void assign_sum(cuda_type *Y,  cuda_type *X1,  
				      cuda_type* X2, const cuda_type alpha) 
{
  unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  Y[idx] = X1[idx] + alpha * X2[idx];
}

// computes y = alpha1 * ( x1 + x2 + beta*x3 )
template <typename cuda_type>
__device__ __forceinline__ void increment_sum_sum(cuda_type *Y,  cuda_type *X1,  cuda_type* X2, 
					    cuda_type* X3, const cuda_type alpha, 
					     const cuda_type beta) 
{
  unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  Y[idx] = alpha*(X1[idx] + X2[idx] + beta*X3[idx]);
}

// computes y = x1 + alpha * x2 ; x2 += x3
template <typename cuda_type>
__device__ __forceinline__ void assign_sum_increment(cuda_type *Y,  cuda_type *X1,  cuda_type* X2, 
					       cuda_type* X3, cuda_type alpha) 
{
  unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  Y[idx] = X1[idx] + alpha*X2[idx];
  X2[idx] += X3[idx];
}


///////////////////////////////////////////////////////////////////////
// hills equation

template <typename cuda_type, typename preprocessor>
struct hills_dynamical_system 
{
  static size_t get_size () {return 2;} 

  __device__ void operator () (cuda_type *S,  cuda_type *D,  cuda_type* O, 
			       cuda_type t, cuda_type max_val, preprocessor prec = preprocessor () )
  {

    const cuda_type nu = 0.08, mR = (1.5 * 0.5);	
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    unsigned int sstride = 6*idx;
    unsigned int ostride = 2*idx;

    cuda_type x = S[sstride];
    cuda_type vx = S[++sstride];
    cuda_type y = S[++sstride];
    cuda_type vy = S[++sstride];
    cuda_type theta = S[++sstride];	
    cuda_type omega = S[++sstride];
	
    cuda_type distance = sqrt(x * x + y * y);

    if(theta < -M_PI) theta += 2 * M_PI;
    if(theta > M_PI) theta -= 2 * M_PI;
	
    cuda_type ul = prec (O[ostride], max_val);
    cuda_type ur = prec (O[++ostride], max_val);
       
    D[sstride] = (ul - ur) * 1/mR;
    D[--sstride] = omega;
    D[--sstride] = -2 * nu * vx + (ul + ur) * sin(theta);
    D[--sstride] = vy;
    D[--sstride] = 2 * nu * vy + 3 * nu * nu * x + (ul + ur) * cos(theta);
    D[--sstride] = vx;
  }
};


/////////////////////////////////////////////////////////////////////////
// runge kutta integrator
//


template <typename cuda_type, typename DynamicalSystem>

__global__ void cu_runge_kutta_integrate (cuda_type  * X , cuda_type * O, cuda_type t , cuda_type dt ,  
					  const cuda_type max_val, const size_t value_size, 
					  const size_t block_size,  DynamicalSystem system = DynamicalSystem())
{
  const size_t datasize = 6;

  cuda_type dxdt [ datasize ] ;

  cuda_type  dh = cuda_type( 0.5 ) * dt;
  cuda_type th = t + dh;

  const cuda_type val2 = cuda_type( 2.0 );

  system(X, dxdt, O, t, max_val);

  cuda_type xt [datasize];

  assign_sum( xt , X , dxdt , dh );

  cuda_type dxt [datasize];

  system( xt , dxt , O, th , max_val);

  assign_sum( xt, X, dxt , dh );

  cuda_type dxm [datasize];
  system( xt , dxm , O, th , max_val);

  assign_sum_increment( xt, X, dxm,dxt, dt );

  system( xt , dxt , O, cuda_type( t + dt ) , max_val);

  increment_sum_sum( X, dxdt, dxt,  dxm, 
		     dt /  cuda_type( 6.0 ) , val2 );

}


template <typename cuda_type, typename dynamicalsystem>
void runge_kutta_integrate (cuda_type  * X , cuda_type * O, cuda_type t , cuda_type dt , cuda_type max_val, size_t value_size, size_t block_size, dim3 g, dim3 b)
  {
    cu_runge_kutta_integrate <cuda_type, dynamicalsystem ><<<g, b>>>(X , O, t , dt, max_val, value_size, block_size);
  } 


template <>
void runge_kutta_integrate <float, hills_dynamical_system <float, scale_functor<float> > > (float  * X , float * O, float t , float dt , 
											    float max_val, size_t value_size, 
											    size_t block_size, dim3 g, dim3 b)
{
    cu_runge_kutta_integrate <float, hills_dynamical_system <float, scale_functor<float> > ><<<g, b>>>(X , O, t , dt, max_val, value_size, block_size);
  } 


template <>
void runge_kutta_integrate <double, hills_dynamical_system <double, scale_functor<double> > > (double  * X , double * O, double t , double dt , 
											    double max_val, size_t value_size, 
											    size_t block_size, dim3 g, dim3 b)
  {
    cu_runge_kutta_integrate <double, hills_dynamical_system <double, scale_functor<double> > ><<<g, b>>>(X , O, t , dt, max_val, value_size, block_size);
  } 


///////////////////////////////////////////////////////////////////////
// fitness kernels


template <typename ty, typename preprocessor>
__global__ void cu_compute_fitness_mindis_kernel(ty *S , ty *O, ty width, 
						 preprocessor prep = preprocessor())
{

  //  unsigned int bx = blockIdx.x, by = blockIdx.y;
  //Todo remove S[1] as its not used
  unsigned int tx = threadIdx.x;
  unsigned int offset = tx * 4;

  for (size_t i = 0; i < 4; ++i)
    {
       S[offset + i] = prep( S[offset + i] );
    }

  ty distance = sqrt(S[offset] * S[offset] + S[offset + 2] * S[offset + 2]);
  ty speed    = sqrt(S[offset + 1] * S[offset + 1] + S[offset + 3] * S[offset + 3]);
  O[tx] =  1/( 1 + distance );
}


template <typename ty, typename preprocessor>
void cu_compute_fitness_mindis(ty *S , ty *O, ty width, dim3 g, dim3 b )
{
  cu_compute_fitness_mindis_kernel<ty, preprocessor><<<g, b>>>(S , O, width);
}

template <>
void cu_compute_fitness_mindis<float, nop_functor<float> >(float *S , float *O, float width, dim3 g, dim3 b )
{
  cu_compute_fitness_mindis_kernel<float, nop_functor<float> ><<<g, b>>>(S , O, width);
}


template <>
void cu_compute_fitness_mindis<double, nop_functor<double> >(double *S , double *O, double width, dim3 g, dim3 b )
{
  cu_compute_fitness_mindis_kernel<double, nop_functor<double> ><<<g, b>>>(S , O, width);
}


/*template <typename ty>
__global__ void cu_compute_fitness_mindis_theta_kernel(ty *S , ty *O , ty width)
{

  //unsigned int bx = blockIdx.x, by = blockIdx.y;
  unsigned int tx = threadIdx.x;
  unsigned int offset = tx * 5;

  ty distance = sqrt(S[offset ] * S[offset] + S[offset + 2] * S[offset + 2]);
  ty speed    = sqrt(S[offset + 1] * S[offset + 1] + S[offset + 3] * S[offset + 3]);		// sqrt(vx^2 + vy^2)
  ty theta = S[offset + 4];
  // keep theta between -180 and +180°

  //Todo optimise
  if(theta < -M_PI) theta += 2 * M_PI;
    if(theta > M_PI) theta -= 2 * M_PI;	
  ty fitness =  1/( (1+distance) * (1+speed) * (1+fabs(theta)) );
}

template <typename ty>
__global__ void cu_compute_fitness_mindis_simple_kernel(ty *S , ty *O , 
							ty * init_distance, ty width)
{
  unsigned int tx = threadIdx.x;
  unsigned int offset = tx * 3;

  ty distance = sqrt(S[offset] * S[offset] + S[offset + 2] * S[offset + 2]);
  O[tx] =  0; 
  if (distance < init_distance[tx])
    {
      O[tx] = 1/( 1 + distance );
    }     
}

template <typename ty>
__global__ void cu_compute_fitness_mindis_noatt_kernel(ty *S , ty *O , 
						       ty * init_distance,ty vic_distance,  
						       ty vic_speed, ty max_dock_time, ty t,
						       ty width)
{

  unsigned int tx = threadIdx.x;
  unsigned int offset = tx * 4;

  ty distance = sqrt(S[offset] * S[offset] + S[offset + 2] * S[offset + 2]);
  ty speed    = sqrt(S[offset + 1] * S[offset + 1] + S[offset + 3] * S[offset + 3]);		// sqrt(vx^2 + vy^2)
  O[tx] = 0.0;
  if(distance < init_distance[tx]/2) {
    O[tx] = 1.0/((1 + distance) * (1 + speed));				
    if(distance < vic_distance && speed < 0.1)
      O[tx] += O[tx] * (max_dock_time - t)/max_dock_time;
  }		
}


template <typename ty>
__global__ void cu_compute_fitness_twodee1_kernel(ty *S , ty *O , 
						  ty * init_distance, ty max_docking_time, ty t,
						  ty width)
{

  unsigned int tx = threadIdx.x;
  unsigned int offset = tx * 5;

  ty distance = sqrt(S[offset] * S[offset] + S[offset + 2] * S[offset + 2]);
  ty speed    = sqrt(S[offset + 1] * S[offset + 1] + S[offset + 3] * S[offset + 3]);		// sqrt(vx^2 + vy^2)
  ty theta = S[offset + 4];
  if(theta < -M_PI) theta += 2 * M_PI;
  if(theta > M_PI) theta -= 2 * M_PI;	
  ty timeBonus = (max_docking_time - t)/max_docking_time;
  O[tx] = 1.0/((1+distance)*(1+fabs(theta))*(speed+1));
  if (init_distance[tx] > distance/2) {
    if(O[tx] > 0.87)
      O[tx] += O[tx] * timeBonus;	
  } 
  else
    O[tx] = 0;	
}

template <typename ty>
__global__ void cu_compute_fitness_twodee2_kernel(ty *S0 , ty *O , 
						  ty *init_distance,ty vic_distance,  
						  ty vic_speed, ty vic_orientation, ty max_dock_time, ty t,
						  ty width)
{

  unsigned int tx = threadIdx.x;
  ty * S = &S0[tx*5];

  ty distance = sqrt(S[0] * S[0] + S[2] * S[2]);
  ty speed    = sqrt(S[1] * S[1] + S[3] * S[3]);		// sqrt(vx^2 + vy^2)
  ty theta = S[4];
  // keep theta between -180 and +180°
  if(theta < -M_PI) theta += 2 * M_PI;
  if(theta > M_PI) theta -= 2 * M_PI;	
  // based on Christos' TwoDee function
  ty timeBonus = (max_dock_time - t)/max_dock_time;
  ty alpha = 1.0/((1+distance)*(1+fabs(theta))*(speed+1));
  if (init_distance[tx] > distance/2) {
    if (distance < vic_distance && fabs(theta) < vic_orientation && speed < vic_speed)
      O[tx] = alpha + alpha * timeBonus;	
    else
      O[tx] = alpha;
  } else
    O[tx] = 0;

}

template <typename ty>
__global__ void cu_compute_fitness_twodee3_kernel(ty *S0 , ty *O , 
						  ty *init_distance,ty vic_distance,  
						  ty vic_speed, ty vic_orientation, ty max_dock_time, ty t,
						  ty width)
{

  unsigned int bx = blockIdx.x, by = blockIdx.y;
  unsigned int tx = threadIdx.x, tty = threadIdx.y;
  ty *  S = &S0[tx*5];

  ty distance = sqrt(S[0] * S[0] + S[2] * S[2]);
  ty speed    = sqrt(S[1] * S[1] + S[3] * S[3]);		// sqrt(vx^2 + vy^2)
  ty theta = S[4];
  // keep theta between -180 and +180°
  if(theta < -M_PI) theta += 2 * M_PI;
  if(theta > M_PI) theta -= 2 * M_PI;	
  // christo's but as soon as we reach the vicinity the
  // individual gets 1.00 as fitness + then the timeBonus
  ty timeBonus = (max_dock_time - t)/max_dock_time;
  ty alpha = 1.0/((1+ distance )*(1+fabs(theta))*(speed+1));
  if (init_distance[tx] > distance / 2) {
    if (distance < vic_distance && fabs(theta) < vic_orientation && speed < vic_speed)
      O[tx] = 1 + timeBonus;	
    else
      O[tx] = alpha;
  } else
    O[tx] = 0;
}*/


/////////////////////////////////////////////////////////////////////////////
//Misc kernels


#endif 
